#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2011, CESNET z.s.p.o
 * Copyright (c) 2011, Silicon Genome, LLC.
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
 
#include "gpujpeg_huffman_gpu_decoder.h"
#include "gpujpeg_format_type.h"
#include "gpujpeg_util.h"

/** Natural order in constant memory */
__constant__ int gpujpeg_huffman_gpu_decoder_order_natural[64];

/**
 * Fill more bit to current get buffer
 * 
 * @param get_bits
 * @param get_buff
 * @param data
 * @param data_size
 * @return void
 */
__device__ inline void
gpujpeg_huffman_gpu_decoder_decode_fill_bit_buffer(int & get_bits, int & get_buff, uint8_t* & data, int & data_size)
{
    while ( get_bits < 25 ) {
        //Are there some data?
        if( data_size > 0 ) { 
            // Attempt to read a byte
            //printf("read byte %X 0x%X\n", (int)data, (unsigned char)*data);
            unsigned char uc = *data++;
            data_size--;            

            // If it's 0xFF, check and discard stuffed zero byte
            if ( uc == 0xFF ) {
                do {
                    //printf("read byte %X 0x%X\n", (int)data, (unsigned char)*data);
                    uc = *data++;
                    data_size--;
                } while ( uc == 0xFF );

                if ( uc == 0 ) {
                    // Found FF/00, which represents an FF data byte
                    uc = 0xFF;
                } else {                
                    // There should be enough bits still left in the data segment;
                    // if so, just break out of the outer while loop.
                    //if (m_nGetBits >= nbits)
                    if ( get_bits >= 0 )
                        break;
                }
            }

            get_buff = (get_buff << 8) | ((int) uc);
            get_bits += 8;            
        }
        else
            break;
    }
}

/**
 * Get bits
 * 
 * @param nbits  Number of bits to get
 * @param get_bits
 * @param get_buff
 * @param data
 * @param data_size
 * @return bits
 */
__device__ inline int
gpujpeg_huffman_gpu_decoder_get_bits(int nbits, int & get_bits, int & get_buff, uint8_t* & data, int & data_size)
{
    //we should read nbits bits to get next data
    if( get_bits < nbits )
        gpujpeg_huffman_gpu_decoder_decode_fill_bit_buffer(get_bits, get_buff, data, data_size);
    get_bits -= nbits;
    return (int)(get_buff >> get_bits) & ((1 << nbits) - 1);
}


/**
 * Special Huffman decode:
 * (1) For codes with length > 8
 * (2) For codes with length < 8 while data is finished
 * 
 * @param table
 * @param min_bits
 * @param get_bits
 * @param get_buff
 * @param data
 * @param data_size
 * @return int
 */
__device__ inline int
gpujpeg_huffman_gpu_decoder_decode_special_decode(struct gpujpeg_table_huffman_decoder* table, int min_bits, int & get_bits, int & get_buff, uint8_t* & data, int & data_size)
{
    // HUFF_DECODE has determined that the code is at least min_bits
    // bits long, so fetch that many bits in one swoop.
    int code = gpujpeg_huffman_gpu_decoder_get_bits(min_bits, get_bits, get_buff, data, data_size);

    // Collect the rest of the Huffman code one bit at a time.
    // This is per Figure F.16 in the JPEG spec.
    int l = min_bits;
    while ( code > table->maxcode[l] ) {
        code <<= 1;
        code |= gpujpeg_huffman_gpu_decoder_get_bits(1, get_bits, get_buff, data, data_size);
        l++;
    }

    // With garbage input we may reach the sentinel value l = 17.
    if ( l > 16 ) {
        // Fake a zero as the safest result
        return 0;
    }
    
    return table->huffval[table->valptr[l] + (int)(code - table->mincode[l])];
}

/**
 * To find dc or ac value according to category and category offset
 * 
 * @param category
 * @param offset
 * @return int
 */
__device__ inline int
gpujpeg_huffman_gpu_decoder_value_from_category(int category, int offset)
{
    // Method 1: 
    // On some machines, a shift and add will be faster than a table lookup.
    // #define HUFF_EXTEND(x,s) \
    // ((x)< (1<<((s)-1)) ? (x) + (((-1)<<(s)) + 1) : (x)) 

    // Method 2: Table lookup
    // If (offset < half[category]), then value is below zero
    // Otherwise, value is above zero, and just the offset 
    // entry n is 2**(n-1)
    const int half[16] =    { 
        0x0000, 0x0001, 0x0002, 0x0004, 0x0008, 0x0010, 0x0020, 0x0040, 
        0x0080, 0x0100, 0x0200, 0x0400, 0x0800, 0x1000, 0x2000, 0x4000
    };

    //start[i] is the starting value in this category; surely it is below zero
    // entry n is (-1 << n) + 1
    const int start[16] = { 
        0, ((-1)<<1) + 1, ((-1)<<2) + 1, ((-1)<<3) + 1, ((-1)<<4) + 1,
        ((-1)<<5) + 1, ((-1)<<6) + 1, ((-1)<<7) + 1, ((-1)<<8) + 1,
        ((-1)<<9) + 1, ((-1)<<10) + 1, ((-1)<<11) + 1, ((-1)<<12) + 1,
        ((-1)<<13) + 1, ((-1)<<14) + 1, ((-1)<<15) + 1 
    };    

    return (offset < half[category]) ? (offset + start[category]) : offset;    
}

/**
 * Get category number for dc, or (0 run length, ac category) for ac.
 * The max length for Huffman codes is 15 bits; so we use 32 bits buffer    
 * m_nGetBuff, with the validated length is m_nGetBits.
 * Usually, more than 95% of the Huffman codes will be 8 or fewer bits long
 * To speed up, we should pay more attention on the codes whose length <= 8
 * 
 * @param table
 * @param get_bits
 * @param get_buff
 * @param data
 * @param data_size
 * @return int
 */
__device__ inline int
gpujpeg_huffman_gpu_decoder_get_category(int & get_bits, int & get_buff, uint8_t* & data, int & data_size, struct gpujpeg_table_huffman_decoder* table)
{
    // If left bits < 8, we should get more data
    if ( get_bits < 8 )
        gpujpeg_huffman_gpu_decoder_decode_fill_bit_buffer(get_bits, get_buff, data, data_size);

    // Call special process if data finished; min bits is 1
    if( get_bits < 8 )
        return gpujpeg_huffman_gpu_decoder_decode_special_decode(table, 1, get_bits, get_buff, data, data_size);

    // Peek the first valid byte    
    int look = ((get_buff >> (get_bits - 8)) & 0xFF);
    int nb = table->look_nbits[look];

    if ( nb ) { 
        get_bits -= nb;
        return table->look_sym[look]; 
    } else {
        //Decode long codes with length >= 9
        return gpujpeg_huffman_gpu_decoder_decode_special_decode(table, 9, get_bits, get_buff, data, data_size);
    }
}

/**
 * Decode one 8x8 block
 *
 * @return 0 if succeeds, otherwise nonzero
 */
__device__ inline int
gpujpeg_huffman_gpu_decoder_decode_block(int & dc, int & get_bits, int & get_buff, uint8_t* & data, int & data_size, int16_t* data_output, 
                                      struct gpujpeg_table_huffman_decoder* table_dc, struct gpujpeg_table_huffman_decoder* table_ac)
{
    // Section F.2.2.1: decode the DC coefficient difference
    // get dc category number, s
    int s = gpujpeg_huffman_gpu_decoder_get_category(get_bits, get_buff, data, data_size, table_dc);
    if ( s ) {
        // Get offset in this dc category
        int r = gpujpeg_huffman_gpu_decoder_get_bits(s, get_bits, get_buff, data, data_size);
        // Get dc difference value
        s = gpujpeg_huffman_gpu_decoder_value_from_category(s, r);
    }

    // Convert DC difference to actual value, update last_dc_val
    s += dc;
    dc = s;

    // Output the DC coefficient (assumes gpujpeg_natural_order[0] = 0)
    data_output[0] = s;
    
    // Section F.2.2.2: decode the AC coefficients
    // Since zeroes are skipped, output area must be cleared beforehand
    for ( int k = 1; k < 64; k++ ) {
        // s: (run, category)
        int s = gpujpeg_huffman_gpu_decoder_get_category(get_bits, get_buff, data, data_size, table_ac);
        // r: run length for ac zero, 0 <= r < 16
        int r = s >> 4;
        // s: category for this non-zero ac
        s &= 15;
        if ( s ) {
            //    k: position for next non-zero ac
            k += r;
            //    r: offset in this ac category
            r = gpujpeg_huffman_gpu_decoder_get_bits(s, get_bits, get_buff, data, data_size);
            //    s: ac value
            s = gpujpeg_huffman_gpu_decoder_value_from_category(s, r);

            data_output[gpujpeg_huffman_gpu_decoder_order_natural[k]] = s;            
        } else {
            // s = 0, means ac value is 0 ? Only if r = 15.  
            //means all the left ac are zero
            if ( r != 15 )
                break;
            k += 15;
        }
    }
    
    /*printf("GPU Decode Block\n");
    for ( int y = 0; y < 8; y++ ) {
        for ( int x = 0; x < 8; x++ ) {
            printf("%4d ", data_output[y * 8 + x]);
        }
        printf("\n");
    }*/
    
    return 0;
}

/**
 * Huffman decoder kernel
 * 
 * @return void
 */
__global__ void
gpujpeg_huffman_decoder_decode_kernel(
    int restart_interval,
    int comp_block_count,
    int comp_segment_count,
    int segment_count,    
    uint8_t* d_data_scan,
    int data_scan_size,
    int* d_data_scan_index,
    int16_t* d_data_decompressed,
    struct gpujpeg_table_huffman_decoder* d_table_y_dc,
    struct gpujpeg_table_huffman_decoder* d_table_y_ac,
    struct gpujpeg_table_huffman_decoder* d_table_cbcr_dc,
    struct gpujpeg_table_huffman_decoder* d_table_cbcr_ac
)
{
    int comp_index = blockIdx.y;
    int comp_segment_index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( comp_segment_index >= comp_segment_count )
        return;
    int segment_index = comp_index * comp_segment_count + comp_segment_index;
    if ( segment_index >= segment_count )
        return;
    
    // Get huffman tables
    struct gpujpeg_table_huffman_decoder* d_table_dc = NULL;
    struct gpujpeg_table_huffman_decoder* d_table_ac = NULL;
    if ( comp_index == 0 ) {
        d_table_dc = d_table_y_dc;
        d_table_ac = d_table_y_ac;
    } else {
        d_table_dc = d_table_cbcr_dc;
        d_table_ac = d_table_cbcr_ac;
    }
    
    // Start coder
    int get_buff = 0;
    int get_bits = 0;
    int dc = 0;
    
    // Prepare data pointer and its size
    int data_index = d_data_scan_index[segment_index];
    uint8_t* data = &d_data_scan[data_index];
    int data_size = 0;
    if ( (segment_index + 1) >= segment_count )
        data_size = data_scan_size - data_index;
    else
        data_size = d_data_scan_index[segment_index + 1] - data_index;
    
    // Encode blocks in restart segment
    int comp_block_index = comp_segment_index * restart_interval;
    for ( int block = 0; block < restart_interval; block++ ) {
        // Skip blocks out of memory
        if ( comp_block_index >= comp_block_count )
            break;
        // Decode block
        int data_index = (comp_block_count * comp_index + comp_block_index) * GPUJPEG_BLOCK_SIZE * GPUJPEG_BLOCK_SIZE;
        gpujpeg_huffman_gpu_decoder_decode_block(
            dc,
            get_bits,
            get_buff,
            data,
            data_size,
            &d_data_decompressed[data_index],
            d_table_dc,
            d_table_ac
        );
        comp_block_index++;
    }
}

/** Documented at declaration */
int
gpujpeg_huffman_gpu_decoder_init()
{
    // Copy natural order to constant device memory
    hipMemcpyToSymbol(HIP_SYMBOL(
        "gpujpeg_huffman_gpu_decoder_order_natural"),
        gpujpeg_order_natural, 
        64 * sizeof(int),
        0,
        hipMemcpyHostToDevice
    );
    gpujpeg_cuda_check_error("Huffman decoder init");
    
    return 0;
}

/** Documented at declaration */
int
gpujpeg_huffman_gpu_decoder_decode(struct gpujpeg_decoder* decoder)
{    
    assert(decoder->restart_interval > 0);
    
    int comp_block_cx = (decoder->param_image.width + GPUJPEG_BLOCK_SIZE - 1) / GPUJPEG_BLOCK_SIZE;
    int comp_block_cy = (decoder->param_image.height + GPUJPEG_BLOCK_SIZE - 1) / GPUJPEG_BLOCK_SIZE;
    int comp_block_count = comp_block_cx * comp_block_cy;
    int comp_segment_count = gpujpeg_div_and_round_up(comp_block_count, decoder->restart_interval);
    
    // Run kernel
    dim3 thread(32);
    dim3 grid(gpujpeg_div_and_round_up(comp_segment_count, thread.x), decoder->param_image.comp_count);
    gpujpeg_huffman_decoder_decode_kernel<<<grid, thread>>>(
        decoder->restart_interval,
        comp_block_count, 
        comp_segment_count,
        decoder->segment_count,
        decoder->d_data_scan,
        decoder->data_scan_size,
        decoder->d_data_scan_index,
        decoder->d_data_quantized,
        decoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_DC],
        decoder->d_table_huffman[GPUJPEG_COMPONENT_LUMINANCE][GPUJPEG_HUFFMAN_AC],
        decoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_DC],
        decoder->d_table_huffman[GPUJPEG_COMPONENT_CHROMINANCE][GPUJPEG_HUFFMAN_AC]
    );
    hipError_t cuerr = hipDeviceSynchronize();
    if ( cuerr != hipSuccess ) {
        fprintf(stderr, "Huffman decoding failed: %s!\n", hipGetErrorString(cuerr));
        return -1;
    }
    
    return 0;
}
